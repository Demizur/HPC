#include "hip/hip_runtime.h"

#include "lodepng.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>

#define BLUR_SIZE 7
#define R 0
#define G 1
#define B 2
#define A 3

__global__ void blur(unsigned char* in, unsigned char* out, int width, int height, int num_channel, int channel, int copy_A) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if(col < width && row < height) {
    int pixVal = 0;
    int pixels = 0;
    if (copy_A)
      out[row*width*num_channel+col*num_channel+A] = in[row*width*num_channel+col*num_channel+A];
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
        int curRow = row + blurRow;
        int curCol = col + blurCol;
        if(curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
          pixVal += in[curRow * width * num_channel + curCol * num_channel + channel];
          pixels++;
        }
      }
    }
    out[row * width * num_channel + col * num_channel + channel] = (unsigned char)(pixVal/pixels);
  }
}

int main() {

  int width, height,n;
  unsigned char *image = stbi_load("image.png",&width,&height,&n,0);

  unsigned char* Input_Image = NULL;
  unsigned char* Output_Image = NULL;
  hipMalloc((void**)&Input_Image, sizeof(unsigned char)* height * width * n);
  hipMalloc((void**)&Output_Image, sizeof(unsigned char)* height * width * n);

  hipMemcpy(Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);

  int BlocksPerThread = 16;
  dim3 blockSize(BlocksPerThread, BlocksPerThread, 1);
  dim3 gridSize(width/blockSize.x, height/blockSize.y,1);

  //blurring
  blur <<<gridSize, blockSize>>>(Input_Image, Output_Image, width, height,n,R,0);
  blur <<<gridSize, blockSize>>>(Input_Image, Output_Image, width, height,n,G,0);
  blur <<<gridSize, blockSize>>>(Input_Image, Output_Image, width, height,n,B,1);
  
  hipDeviceSynchronize();

  hipMemcpy(image, Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
  hipFree(Input_Image);
  hipFree(Output_Image);
  stbi_write_png("blurred.png", width, height, n, image, width * n);

  return 0;
}