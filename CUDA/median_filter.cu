#include "hip/hip_runtime.h"

#include "lodepng.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>

#define WINDOW_SIZE 3
#define MEDIAN_DIMENSION 3    
#define MEDIAN_LENGTH 9   
#define BLOCK_WIDTH 16  
#define BLOCK_HEIGHT 16

#define R 0
#define G 1
#define B 2
#define A 3

__global__ void median( unsigned char *image, int width,int height, int num_channel, int channel, int copy_A){

    unsigned char out[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH];

    int iterator;
    int Half_Of_MEDIAN_LENGTH =(MEDIAN_LENGTH/2)+1;
    int start=MEDIAN_DIMENSION/2;
    int end=start+1;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y*blockDim.y+threadIdx.x;   

      if(x>=width || y>=height)
        return;
        if (copy_A)
        *(image+(y*width * num_channel)+x * num_channel + A) = image[y*width*num_channel+x*num_channel+A];
            if (x == 0 || x == width - start || y == 0
                || y == height - start) {
            } else {             
                iterator = 0;
                for (int r = x - start; r < x + (end); r++) {
                    for (int c = y - start; c < y + (end); c++) {
                        out[tid][iterator] =*(image+(c*width*num_channel)+r * num_channel + channel);
                        iterator++;
                    }
                }
                      int t,j,i;
                    for ( i = 1 ; i< MEDIAN_LENGTH ; i++) {
                        j = i;
                        while ( j > 0 && out[tid][j] < out[tid][j-1]) {
                            t= out[tid][j];
                            out[tid][j]= out[tid][j-1];
                            out[tid][j-1] = t;
                            j--;
                        }
                    }

                    *(image+(y*width * num_channel)+x * num_channel + channel)= out[tid][Half_Of_MEDIAN_LENGTH-1];
            }  
}

int main() {

  int width, height,n;
  unsigned char *image = stbi_load("image.png",&width,&height,&n,0);

  unsigned char* Input_Image = NULL;
  unsigned char* Output_Image = NULL;
  hipMalloc((void**)&Input_Image, sizeof(unsigned char)* height * width * n);
  hipMalloc((void**)&Output_Image, sizeof(unsigned char)* height * width * n);

  hipMemcpy(Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);

  int BlocksPerThread = 16;
  dim3 blockSize(BlocksPerThread, BlocksPerThread, 1);
  dim3 gridSize(width/blockSize.x, height/blockSize.y,1);

  //median filtering
  median <<<gridSize, blockSize>>>(Input_Image, width, height,n,R,0);
  median <<<gridSize, blockSize>>>(Input_Image, width, height,n,G,0);
  median <<<gridSize, blockSize>>>(Input_Image, width, height,n,B,1);
  
  hipDeviceSynchronize();

  hipMemcpy(image, Input_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
  hipFree(Input_Image);
  hipFree(Output_Image);
  stbi_write_png("median.png", width, height, n, image, width * n);

  return 0;
}