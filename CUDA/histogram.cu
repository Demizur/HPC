#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

__global__ void Hist(int height, int width,  uint8_t *d_image, unsigned int *d_res)
{
        int globalidx = threadIdx.x + blockDim.x * blockIdx.x;
	long int size = height * width;	
	// uint8_t tid = threadIdx.x;

	if (globalidx < size)
	{

		unsigned char value = d_image[globalidx];
		int bin = value % 256;
		atomicAdd(&d_res[bin], 1);
		//__syncthreads();				
	}
}

int main(int argc, char **argv)
{
	int width, height, bpp, size;
	FILE *fp;
	fp = fopen("hist.txt", "w");

	uint8_t* h_image_init = stbi_load("grey_cat.jpg", &width, &height, &bpp, 3);		
	size = height * width;

	uint8_t* h_image = (uint8_t *) malloc(sizeof(uint8_t) * size);
	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < height; j++)
		{
			h_image[j*width + i] = (h_image_init[j*width*3 + i*3] + \
						h_image_init[j*width*3 + i*3 + 1] + \
						h_image_init[j*width*3 + i*3 + 2]) / 3.;		
		}
	}

	uint8_t *d_image;
	unsigned int *d_res;
	unsigned int *h_res = (unsigned int *) malloc(sizeof(unsigned int) * 256);
	hipMalloc(&d_image, sizeof(uint8_t) * size);
	hipMalloc(&d_res, sizeof(unsigned int) * 256);
	hipMemset(d_res, 0, sizeof(unsigned int) * 256);
	
	hipMemcpy(d_image, h_image, sizeof(uint8_t) * size, hipMemcpyHostToDevice);
  int block_size, grid_size;
	block_size = 256;
  grid_size = size / 256;

  dim3 dimBlock(block_size);
  dim3 dimGrid(grid_size);

	Hist<<<dimGrid, dimBlock>>>(height, width, d_image, d_res);
	hipDeviceSynchronize();	

	hipMemcpy(h_res, d_res, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < 256; i++)
	{
		fprintf(fp, "%d\t", h_res[i]);
	}
	fprintf(fp, "\n");

	free(h_image);
	free(h_res);
	hipFree(d_image);
	hipFree(d_res);
	fclose(fp);

	return 0;	
}