
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Init(int n, int *d_hist)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx < n) d_hist[globalIdx] = 0;
}

__global__ void Histogram(int *d_hist, int *d_image, long int i)
{
    long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("hist:%d\n", blockIdx.x);//, blockDim.x, threadIdx.x);
    int tmp;

    tmp = d_image[idx];
    atomicAdd(&(d_hist[tmp]), 1);
    __syncthreads();
}

int main(int ac, char **av)
{
    int height = 441;
    int width = 350;
 
    int *h_image = (int*)malloc(sizeof(int) * height * width);
    int *h_hist = (int *)malloc(sizeof(int)*256);

 
    FILE *fp;
    int ch;
    fp = fopen("greysacaleimage.csv", "r");
    long int i = 0;
    int tmp = 0;
    while (!feof(fp) && !ferror(fp))
    {
        ch = getc(fp);
        if (ch != EOF)
        {
            if (ch < 32)
              tmp = 0;
            else if (ch != 32)
            {
                if (tmp == 0) tmp += ch - 48;
                else if (tmp != 0) tmp = tmp * 10 + ch - 48;
            }
            else if(ch == 32)
            {
                h_image[i] = tmp;
                tmp = 0;
                i++;
            }
        }
    }
    printf("\n");
    fclose(fp);

    int *d_image;
    int *d_hist;
  
    hipMalloc(&d_image, sizeof(int) * height * width);
    hipMalloc(&d_hist, sizeof(int)*256);

    hipMemcpy(d_image, h_image, sizeof(int)*width*height, hipMemcpyHostToDevice);
    hipMemcpy(d_hist, h_hist, sizeof(int)*256, hipMemcpyHostToDevice);
 
    Init<<<1, 256>>>(256, d_hist);
    Histogram<<<1058, 1024>>>(d_hist, d_image, i);
 
    hipMemcpy(h_image, d_image, sizeof(int)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_hist, d_hist, sizeof(float)*256, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    FILE *hist;
    hist = fopen("histogram.csv","w");
    for (int i = 0; i < 256; i++) fprintf(hist, "%d ", h_hist[i]);
 
    free(h_image);
    free(h_hist);
    hipFree(d_image);
    hipFree(d_hist);

    return(0);
}