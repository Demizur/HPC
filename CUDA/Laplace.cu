
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 50;
#define max_iter 500

__global__ void Init(float * dA)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < N*N)
    { 
        if (blockIdx.x != 0) dA[idx] = 0;
        else 
        {
            if ((threadIdx.x == 0) | (threadIdx.x == N-1)) dA[idx] = 0;
            else dA[idx] = 1;
        }
    }  
}

__global__ void Step(float *T_old, float *T_new)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int up, down, right, left;

    if(idx < N*N)
    { 
        if((blockIdx.x != 0) && (blockIdx.x !=  N-1))
        {
            if((threadIdx.x != 0) && (threadIdx.x !=  N-1))
            {
                right = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
                left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;            
                up = blockIdx.x * blockDim.x + threadIdx.x + 1;
                down = blockIdx.x * blockDim.x + threadIdx.x - 1;
             
                T_new[idx] = 0.25* (T_old[up] + T_old[down] + T_old[right] + T_old[left]);
            }
        }
    }  
}
int main()
{
    float *T_old;
    float *T;

    hipMallocManaged(&T_old, sizeof(float)*N*N);
    hipMallocManaged(&T, sizeof(float)*N*N);

    Init<<<N,N>>>(T_old);
    Init<<<N,N>>>(T);
 
    int k = 0;
    while(k< max_iter){
        Step<<<N,N>>>(T_old, T);
        Step<<<N,N>>>(T, T_old);
        k++;
    }

    hipDeviceSynchronize();

    FILE *fp=NULL;
    fp = fopen("heatmap.txt","w");

    for (int j=0; j < N; j++)
    {
        for (int i=0; i < N; i++)
        {
            fprintf(fp, "%f ", T[i + j * N]);
        }
        fprintf(fp, "\n");
    }

    hipDeviceSynchronize();
 
    fclose(fp);
    hipFree(T_old);
    hipFree(T);

    return 0;
}